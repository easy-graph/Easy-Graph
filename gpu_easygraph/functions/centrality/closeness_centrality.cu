#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#include "common.h"

namespace gpu_easygraph {

static __device__ double atomicAddDouble (
    _OUT_ double* address, 
    _IN_ double val
)
{
	unsigned long long int* address_as_ull =
		(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val +
			__longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}



static __device__ double atomicMinDouble (
    _OUT_ double *address, 
    _IN_ double val
)
{
	unsigned long long ret = __double_as_longlong(*address);
	while (val < __longlong_as_double(ret))
	{
		unsigned long long old = ret;
		if ((ret = atomicCAS((unsigned long long *)address, old, __double_as_longlong(val))) == old)
			break;
	}
	return __longlong_as_double(ret);
}



static __global__ void d_calc_min_edge (
    _IN_ int* d_V,
    _IN_ int* d_E,
    _IN_ double* d_W,
    _IN_ int len_V,
    _IN_ int len_E,
    _OUT_ double* d_min_edge
)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tnum = blockDim.x * gridDim.x;

    for (int u = tid; u < len_V; u += tnum) {
		double curr_min = EG_DOUBLE_INF;
        int edge_start = d_V[u];
        int edge_end = d_V[u + 1];
		for(int v = edge_start; v < edge_end; ++v) {
            curr_min = min(curr_min, d_W[v]);
		}
		d_min_edge[u] = curr_min;
    }
}

static __global__ void d_dijkstra_cc (
    _IN_ int* d_V,
    _IN_ int* d_E,
    _IN_ double* d_W,
    _IN_ double* d_min_edge,
    _IN_ int* d_sources,
    _BUFFER_ double* d_dist_2D,
    _BUFFER_ int* d_U_2D,
    _BUFFER_ int* d_F_2D,
    _IN_ int len_V,
    _IN_ int len_E,
    _IN_ int len_sources,
    _IN_ int warp_size,
    _OUT_ double* d_CC
)
{
    for (int s_idx = blockIdx.x; s_idx < len_sources; s_idx += gridDim.x) {
        int s = d_sources[s_idx];

        int* d_U = d_U_2D + blockIdx.x * len_V;
        int* d_F = d_F_2D + blockIdx.x * len_V;
        double* d_dist = d_dist_2D + blockIdx.x * len_V;

        __shared__ int len_F;
        __shared__ double delta;
        __shared__ double dist_accum;
        __shared__ int reachable_cnt;

        for (int i = threadIdx.x; i < len_V; i += blockDim.x) {
            d_U[i] = 1;
            d_dist[i] = EG_DOUBLE_INF;
        }
        __syncthreads();

        if (threadIdx.x == 0) {
            d_dist[s] = 0.0;
            d_F[0] = s;
            len_F = 1;
            delta = 0.0;
            dist_accum = 0.0;
            reachable_cnt = 0;
        }
        __syncthreads();

        while (delta < EG_DOUBLE_INF) {
            for (int j = threadIdx.x; j < len_F * warp_size; j += blockDim.x) {
                int f = d_F[j / warp_size];
                int edge_start = d_V[f];
                int edge_end = d_V[f + 1];
                double dist = d_dist[f];
                for (int e = j % warp_size; e < edge_end - edge_start; e += warp_size) {
                    int adj = d_E[e + edge_start];
                    double relax_w = dist + d_W[e + edge_start];
                    atomicMinDouble(d_dist + adj, relax_w);
                }
                __threadfence_block();
            }
            __syncthreads();

            if (threadIdx.x == 0) {
                delta = EG_DOUBLE_INF;
            }
            __syncthreads();

            for (int i = threadIdx.x; i < len_V; i += blockDim.x) {
                double dist_i = d_dist[i];
                if (d_U[i] == 1 && dist_i < EG_DOUBLE_INF) {
                    atomicMinDouble(&delta, dist_i + d_min_edge[i]);
                }
            }
            __syncthreads();

            if (threadIdx.x == 0) {
				len_F = 0;
			}
			__syncthreads();

            for (int i = threadIdx.x; i < len_V; i += blockDim.x) {
                double dist_i = d_dist[i];
                if (d_U[i] && dist_i <= delta && dist_i < EG_DOUBLE_INF) {
                    d_U[i] = 0;
                    int f_idx = atomicAdd(&len_F, 1);
                    d_F[f_idx] = i;

                    atomicAdd(&reachable_cnt, 1);
                    atomicAddDouble(&dist_accum, d_dist[i]);
                }
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            d_CC[s_idx] = dist_accum == 0.0 ? 0.0 :
                                (double)(reachable_cnt - 1) * 
                                (double)(reachable_cnt - 1) /
                                ((len_V - 1) * dist_accum);
        }
        __syncthreads();
    }
}



// we here use CSR to represent a graph
int cuda_closeness_centrality (
    _IN_ int* V,
    _IN_ int* E,
    _IN_ double* W,
    _IN_ int* sources,
    _IN_ int len_V,
    _IN_ int len_E,
    _IN_ int len_sources,
    _IN_ int warp_size,
    _OUT_ double* CC
)
{
    int cuda_ret = hipSuccess;
    int EG_ret = EG_GPU_SUCC;

    int min_edge_block_size;
    int min_edge_grid_size;
    int dijkstra_block_size;
    int dijkstra_grid_size;

    hipOccupancyMaxPotentialBlockSize(&min_edge_grid_size, &min_edge_block_size, d_calc_min_edge, 0, 0); 
    hipOccupancyMaxPotentialBlockSize(&dijkstra_grid_size, &dijkstra_block_size, d_dijkstra_cc, 0, 0); 

    int *d_V = NULL, *d_E = NULL, *d_sources= NULL;
    int *d_U_2D = NULL, *d_F_2D = NULL;
    double *d_W = NULL, *d_min_edge = NULL, *d_dist_2D = NULL, *d_CC = NULL;

    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_V, sizeof(int) * (len_V + 1)));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_E, sizeof(int) * len_E));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_sources, sizeof(int) * len_sources));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_U_2D, sizeof(int) * dijkstra_grid_size * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_F_2D, sizeof(int) * dijkstra_grid_size * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_W, sizeof(double) * len_E));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_min_edge, sizeof(double) * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_dist_2D, sizeof(double) * dijkstra_grid_size * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_CC, sizeof(double) * len_V));

    EXIT_IF_CUDA_FAILED(hipMemcpy(d_V, V, sizeof(int) * (len_V + 1), hipMemcpyHostToDevice));
    EXIT_IF_CUDA_FAILED(hipMemcpy(d_E, E, sizeof(int) * len_E, hipMemcpyHostToDevice));
    EXIT_IF_CUDA_FAILED(hipMemcpy(d_sources, sources, sizeof(int) * len_sources, hipMemcpyHostToDevice));
    EXIT_IF_CUDA_FAILED(hipMemcpy(d_W, W, sizeof(double) * len_E, hipMemcpyHostToDevice));

    d_calc_min_edge<<<dijkstra_grid_size, dijkstra_block_size>>>(d_V, d_E, d_W, len_V, len_E, d_min_edge);

    d_dijkstra_cc<<<min_edge_grid_size, min_edge_block_size>>>(d_V, d_E, d_W, d_min_edge, d_sources, 
                                    d_dist_2D, d_U_2D, d_F_2D, len_V, len_E, len_sources, warp_size, d_CC);

    EXIT_IF_CUDA_FAILED(hipMemcpy(CC, d_CC, sizeof(double) * len_V, hipMemcpyDeviceToHost));

exit:
    hipFree(d_V);
    hipFree(d_E);
    hipFree(d_sources);
    hipFree(d_U_2D);
    hipFree(d_F_2D);
    hipFree(d_W);
    hipFree(d_min_edge);
    hipFree(d_dist_2D);
    hipFree(d_CC);

    if (cuda_ret != hipSuccess) {
        switch (cuda_ret) {
            case hipErrorOutOfMemory:
                EG_ret = EG_GPU_FAILED_TO_ALLOCATE_DEVICE_MEM;
                break;
            default:
                EG_ret = EG_GPU_DEVICE_ERR;
                break;
        }
    }

    return EG_ret;
}

} // namespace gpu_easygraph