#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#include "common.h"

static __device__ double atomicAddDouble (
    _OUT_ double* address, 
    _IN_ double val
)
{
	unsigned long long int* address_as_ull =
		(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val +
			__longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}



static __device__ double atomicMinDouble (
    _OUT_ double *address, 
    _IN_ double val
)
{
	unsigned long long ret = __double_as_longlong(*address);
	while (val < __longlong_as_double(ret))
	{
		unsigned long long old = ret;
		if ((ret = atomicCAS((unsigned long long *)address, old, __double_as_longlong(val))) == old)
			break;
	}
	return __longlong_as_double(ret);
}



static __global__ void d_calc_min_edge (
    _IN_ int* d_V,
    _IN_ int* d_E,
    _IN_ double* d_W,
    _IN_ int len_V,
    _IN_ int len_E,
    _OUT_ double* d_min_edge
)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < len_V) {
		double curr_min = EG_DOUBLE_INF;
        int edge_start = d_V[tid];
        int edge_end = tid + 1 < len_V ? d_V[tid + 1] : len_E;
		for(int i = edge_start; i < edge_end; ++i) {
            curr_min = min(curr_min, d_W[i]);
		}
		d_min_edge[tid] = curr_min;
	}
}

static __global__ void d_dijkstra_cc (
    _IN_ int* d_V,
    _IN_ int* d_E,
    _IN_ double* d_W,
    _IN_ double* d_min_edge,
    _IN_ int* d_sources,
    _BUFFER_ double* d_dist_2D,
    _BUFFER_ int* d_U_2D,
    _BUFFER_ int* d_F_2D,
    _IN_ int len_V,
    _IN_ int len_E,
    _IN_ int len_sources,
    _IN_ int warp_size,
    _OUT_ double* d_CC
)
{
    for (int s_idx = blockIdx.x; s_idx < len_sources; s_idx += gridDim.x) {
        int s = d_sources[s_idx];

        int* d_U = d_U_2D + blockIdx.x * len_V;
        int* d_F = d_F_2D + blockIdx.x * len_V;
        double* d_dist = d_dist_2D + blockIdx.x * len_V;

        __shared__ int len_F;
        __shared__ double delta;
        __shared__ double dist_accum;
        __shared__ int reachable_cnt;

        for (int i = threadIdx.x; i < len_V; i += blockDim.x) {
            d_U[i] = 1;
            d_dist[i] = EG_DOUBLE_INF;
        }
        __syncthreads();

        if (threadIdx.x == 0) {
            d_dist[s] = 0.0;
            d_F[0] = s;
            len_F = 1;
            delta = 0.0;
            dist_accum = 0.0;
            reachable_cnt = 0;
        }
        __syncthreads();

        while (delta < EG_DOUBLE_INF) {
            for (int j = threadIdx.x; j < len_F * warp_size; j += blockDim.x) {
                int f = d_F[j / warp_size];
                int edge_start = d_V[f];
                int edge_end = f + 1 < len_V ? d_V[f + 1] : len_E;
                double dist = d_dist[f];
                for (int e = j % warp_size; e < edge_end - edge_start; e += warp_size) {
                    int adj = d_E[e + edge_start];
                    double relax_w = dist + d_W[e + edge_start];
                    atomicMinDouble(d_dist + adj, relax_w);
                }
                __threadfence_block();
            }
            __syncthreads();

            if (threadIdx.x == 0) {
                delta = EG_DOUBLE_INF;
            }
            __syncthreads();

            for (int i = threadIdx.x; i < len_V; i += blockDim.x) {
                double dist_i = d_dist[i];
                if (d_U[i] == 1 && dist_i < EG_DOUBLE_INF) {
                    atomicMinDouble(&delta, dist_i + d_min_edge[i]);
                }
            }
            __syncthreads();

            if (threadIdx.x == 0) {
				len_F = 0;
			}
			__syncthreads();

            for (int i = threadIdx.x; i < len_V; i += blockDim.x) {
                double dist_i = d_dist[i];
                if (d_U[i] && dist_i <= delta && dist_i < EG_DOUBLE_INF) {
                    d_U[i] = 0;
                    int f_idx = atomicAdd(&len_F, 1);
                    d_F[f_idx] = i;

                    atomicAdd(&reachable_cnt, 1);
                    atomicAddDouble(&dist_accum, d_dist[i]);
                }
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            d_CC[s_idx] = dist_accum == 0.0 ? 0.0 :
                                (double)(reachable_cnt - 1) * 
                                (double)(reachable_cnt - 1) /
                                ((len_V - 1) * dist_accum);
        }
        __syncthreads();
    }
}



// we here use CSR to represent a graph
int cuda_closeness_centrality (
    _IN_ int* V,
    _IN_ int* E,
    _IN_ double* W,
    _IN_ int* sources,
    _IN_ int len_V,
    _IN_ int len_E,
    _IN_ int len_sources,
    _IN_ int warp_size,
    _OUT_ double* CC
)
{
    int cuda_ret = hipSuccess;
    int EG_ret = EG_GPU_SUCC;

    int block_size = 256;
    size_t grid_size = len_V / block_size + (len_V % block_size != 0);
    size_t mem_free = 0, mem_total = 0;

    int *d_V = NULL, *d_E = NULL, *d_sources= NULL;
    int *d_U_2D = NULL, *d_F_2D = NULL;
    double *d_W = NULL, *d_min_edge = NULL, *d_dist_2D = NULL, *d_CC = NULL;

    EXIT_IF_CUDA_FAILED(hipMemGetInfo(&mem_free, &mem_total));
    while (true) {
        size_t mem_needed = sizeof(int) * len_V // d_V
                        + sizeof(int) * len_E // d_E
                        + sizeof(int) * len_sources // d_sources
                        + sizeof(int) * grid_size * len_V // d_U_2D
                        + sizeof(int) * grid_size * len_V // d_F_2D
                        + sizeof(double) * len_E // d_W
                        + sizeof(double) * len_V // d_min_edge
                        + sizeof(double) * grid_size * len_V // d_dist_2D
                        + sizeof(double) * len_V // d_BC
                        ;
        if (mem_needed < mem_free / 2) {
            break;
        } else {
            grid_size /= 2;
        }
    }

    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_V, sizeof(int) * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_E, sizeof(int) * len_E));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_sources, sizeof(int) * len_sources));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_U_2D, sizeof(int) * grid_size * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_F_2D, sizeof(int) * grid_size * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_W, sizeof(double) * len_E));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_min_edge, sizeof(double) * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_dist_2D, sizeof(double) * grid_size * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_CC, sizeof(double) * len_V));

    EXIT_IF_CUDA_FAILED(hipMemcpy(d_V, V, sizeof(int) * len_V, hipMemcpyHostToDevice));
    EXIT_IF_CUDA_FAILED(hipMemcpy(d_E, E, sizeof(int) * len_E, hipMemcpyHostToDevice));
    EXIT_IF_CUDA_FAILED(hipMemcpy(d_sources, sources, sizeof(int) * len_sources, hipMemcpyHostToDevice));
    EXIT_IF_CUDA_FAILED(hipMemcpy(d_W, W, sizeof(double) * len_E, hipMemcpyHostToDevice));

    d_calc_min_edge<<<grid_size, block_size>>>(d_V, d_E, d_W, len_V, len_E, d_min_edge);

    d_dijkstra_cc<<<grid_size, block_size>>>(d_V, d_E, d_W, d_min_edge, d_sources, d_dist_2D,
                                    d_U_2D, d_F_2D, len_V, len_E, len_sources, warp_size, d_CC);

    EXIT_IF_CUDA_FAILED(hipMemcpy(CC, d_CC, sizeof(double) * len_V, hipMemcpyDeviceToHost));

exit:
    hipFree(d_V);
    hipFree(d_E);
    hipFree(d_sources);
    hipFree(d_U_2D);
    hipFree(d_F_2D);
    hipFree(d_W);
    hipFree(d_min_edge);
    hipFree(d_dist_2D);
    hipFree(d_CC);

    if (cuda_ret != hipSuccess) {
        switch (cuda_ret) {
            case hipErrorOutOfMemory:
                EG_ret = EG_GPU_FAILED_TO_ALLOCATE_DEVICE_MEM;
                break;
            default:
                EG_ret = EG_GPU_DEVICE_ERR;
                break;
        }
    }

    return EG_ret;
}