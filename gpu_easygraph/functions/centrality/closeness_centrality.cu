#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#include "common.h"
#include "utils.h"

__device__ __forceinline__ float atomicMinFloat (
    _OUT_ float * addr,
    _IN_ float value
)
{
    float old;
    old = (value >= 0) ? __int_as_float(atomicMin((int *)addr, __float_as_int(value))) :
        __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value)));

    return old;
}

static __global__ void init_cuda_arr (
    _OUT_ int32_t* d_arr, 
    _IN_ int32_t val, 
    _IN_ int32_t len, 
    _IN_ int32_t source, 
    _IN_ int32_t source_val
)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < len) {
        d_arr[tid] = val;
    }
    if (tid == source) {
        d_arr[tid] = source_val;
    }
}

static __global__ void init_cuda_arr (
    _OUT_ float* d_in, 
    _IN_ float val, 
    _IN_ int32_t len, 
    _IN_ int32_t source, 
    _IN_ float source_val
)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < len) {
        d_in[tid] = val;
    }
    if (tid == source) {
        d_in[tid] = source_val;
    }
}

static __global__ void set_cuda_arr (
    _OUT_ float* d_arr, 
    _IN_ int len, 
    _IN_ float val
)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < len) {
        d_arr[tid] = val;
    }
}

static __global__ void relax (
    _IN_ int32_t* d_V,
    _IN_ int32_t* d_E,
    _IN_ float* d_W,
    _IN_ int32_t len_V,
    _IN_ int32_t len_E,
    _IN_ int32_t* d_U,
    _IN_ int32_t* d_F,
    _OUT_ float* d_delta
)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < len_V && d_F[tid]) {
        int edge_start = d_V[tid];
        int edge_end = tid + 1 < len_V ? d_V[tid + 1] : len_E;
        for (int i = edge_start; i < edge_end; ++i) {
            if (d_U[d_E[i]] && d_delta[d_E[i]] > d_delta[tid] + d_W[i]) {
                atomicMinFloat(d_delta + d_E[i], d_delta[tid] + d_W[i]);
            }
        }
    }
}

static __global__ void min_delta_demarcation (
    _IN_ int32_t* d_V, 
    _IN_ int32_t* d_E, 
    _IN_ float* d_W,
    _IN_ int32_t len_V, 
    _IN_ int32_t len_E,
    _IN_ int32_t* d_U, 
    _IN_ int32_t* d_F, 
    _IN_ float* d_delta,
    _OUT_ float *min_delta
)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= len_V) {
        return;
    }

    if (d_U[tid]) {
        float curr_min = EG_FLOAT_INF;

        int edge_start = d_V[tid];
        int edge_end = tid + 1 < len_V ? d_V[tid + 1] : len_E;
        for (int i = edge_start; i < edge_end; ++i) {
            curr_min = min(curr_min, d_delta[tid] + d_W[i]);
        }

        if (curr_min < *min_delta) {
            atomicMinFloat(min_delta, curr_min);
        }
    }
}

static __global__ void update (
    _IN_ int32_t len_V, 
    _OUT_ int32_t* d_U, 
    _OUT_ int32_t* d_F, 
    _IN_ float* d_delta,
    _IN_ float* d_min_delta
)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= len_V) {
        return;
    }

    if (d_U[tid] && d_delta[tid] <= *d_min_delta) {
        d_U[tid] = 0;
        d_F[tid] = 1;
    }
}

static int dijkstra (
    _IN_ int32_t* d_V, 
    _IN_ int32_t* d_E, 
    _IN_ float* d_W, 
    _IN_ int32_t len_V, 
    _IN_ int32_t len_E, 
    _IN_ int32_t source, 
    _OUT_ float* d_delta
)
{
    int cuda_ret = hipSuccess;

    // define vars
    // d_U means unsettled, d_F means frontier
    int32_t *d_U, *d_F;
    int32_t block_size = 512;
    int32_t grid_size = (len_V + block_size) / block_size;
    float *d_min_delta;
    float h_min_delta;
    const float float_max_inst = EG_FLOAT_INF;

    // initialize vars
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_U, sizeof(int32_t) * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_F, sizeof(int32_t) * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_min_delta, sizeof(float)));
    init_cuda_arr<<<grid_size, block_size>>>(d_U, 1, len_V, source, 0);
    init_cuda_arr<<<grid_size, block_size>>>(d_F, 0, len_V, source, 1);
    init_cuda_arr<<<grid_size, block_size>>>(d_delta, EG_FLOAT_INF, len_V, source, 0.0f);

    // relax, get min, and update
    while (1) {
        grid_size = (len_V + block_size) / block_size;
        relax<<<grid_size, block_size>>>(d_V, d_E, d_W, len_V, len_E, d_U, d_F, d_delta);

        EXIT_IF_CUDA_FAILED(hipMemcpy(d_min_delta, &float_max_inst, sizeof(float), hipMemcpyHostToDevice));
        min_delta_demarcation<<<grid_size, block_size>>>(d_V, d_E, d_W, len_V,
                                                    len_E, d_U, d_F, d_delta, d_min_delta);

        update<<<grid_size, block_size>>>(len_V, d_U, d_F, d_delta, d_min_delta);

        EXIT_IF_CUDA_FAILED(hipMemcpy(&h_min_delta, d_min_delta, sizeof(float), hipMemcpyDeviceToHost));

        if (IS_EQUAL(h_min_delta, EG_FLOAT_INF)) {
            break;
        }
    }
exit:
    hipFree(d_U);
    hipFree(d_F);
    hipFree(d_min_delta);
    return cuda_ret;
}

// All-Pairs Shortest Path
static int APSP (
    _IN_ int32_t* V, 
    _IN_ int32_t* E, 
    _IN_ float* W, 
    _IN_ int32_t* sources,
    _IN_ int32_t len_V, 
    _IN_ int32_t len_E,
    _OUT_ float_t* d_apsp
)
{
    int cuda_ret = hipSuccess;

    int32_t *d_V = NULL, *d_E = NULL;
    float *d_W = NULL;
    
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_V, sizeof(int32_t) * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_E, sizeof(int32_t) * len_E));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_W, sizeof(float) * len_E));

    EXIT_IF_CUDA_FAILED(hipMemcpy(d_V, V, sizeof(int32_t) * len_V, hipMemcpyHostToDevice));
    EXIT_IF_CUDA_FAILED(hipMemcpy(d_E, E, sizeof(int32_t) * len_E, hipMemcpyHostToDevice));
    EXIT_IF_CUDA_FAILED(hipMemcpy(d_W, W, sizeof(float) * len_E, hipMemcpyHostToDevice));
    
    for (int s = 0; s < len_V; ++s) {
        if (sources[s] == EG_GPU_NODE_ACTIVE) {
            EXIT_IF_CUDA_FAILED(dijkstra(d_V, d_E, d_W, len_V, len_E, s, d_apsp + s * len_V));
        }
    }

exit:
    hipFree(d_V);
    hipFree(d_E);
    hipFree(d_W);
    return cuda_ret;
}

static __global__ void calc_CC (
    _IN_ float* d_apsp, 
    _IN_ int32_t* d_sources,
    _IN_ int32_t len_V, 
    _OUT_ float* d_CC
)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= len_V || d_sources[tid] == EG_GPU_NODE_INACTIVE) {
        return;
    }

    int32_t conn_cnt = 0;
    float accum = 0.0f;
    for (int i = 0; i < len_V; ++i) {
        float dist = d_apsp[tid * len_V + i];
        if (!IS_EQUAL(dist, EG_FLOAT_INF)) {
            accum += dist;
            ++conn_cnt;
        }
    }
    d_CC[tid] = IS_EQUAL(accum, 0.0f) ? 0.0f :
            (conn_cnt - 1) * (conn_cnt - 1) / ((len_V - 1) * accum);
}

// we here use CSR to represent a graph
int cuda_closeness_centrality (
    _IN_ int32_t* V, 
    _IN_ int32_t* E, 
    _IN_ float* W, 
    _IN_ int32_t* sources,
    _IN_ int32_t len_V, 
    _IN_ int32_t len_E,  
    _OUT_ float* CC
)
{
    int cuda_ret = hipSuccess;
    int EG_ret = EG_GPU_SUCC;
    int32_t block_size = 512;
    int32_t grid_size = (len_V + block_size) / block_size;
    float* d_apsp = NULL; // a 2d arr arranged in 1d
    float* d_CC = NULL;
    int* d_sources = NULL;

    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_apsp, sizeof(float*) * len_V * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_CC, sizeof(float*) * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_sources, sizeof(int*) * len_V));

    set_cuda_arr<<<grid_size, block_size>>>(d_apsp, len_V * len_V, EG_FLOAT_INF);
    EXIT_IF_CUDA_FAILED(hipMemcpy(d_sources, sources, sizeof(int32_t) * len_V, hipMemcpyHostToDevice));

    EXIT_IF_CUDA_FAILED(APSP(V, E, W, sources, len_V, len_E, d_apsp));

    calc_CC<<<grid_size, block_size>>>(d_apsp, d_sources, len_V, d_CC);

    EXIT_IF_CUDA_FAILED(hipMemcpy(CC, d_CC, sizeof(float) * len_V, hipMemcpyDeviceToHost));

exit:
    hipFree(d_apsp);
    hipFree(d_CC);
    hipFree(d_sources);

    if (cuda_ret != hipSuccess) {
        switch (cuda_ret) {
            case hipErrorOutOfMemory:
                EG_ret = EG_GPU_FAILED_TO_ALLOCATE_DEVICE_MEM;
                break;
            default:
                EG_ret = EG_GPU_DEVICE_ERR;
                break;
        }
    }

    return EG_ret;
}