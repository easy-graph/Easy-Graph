#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#include "common.h"

namespace gpu_easygraph {

static __global__ void d_calc_deg(
    _IN_ int* d_V,
    _IN_ int* d_E,
    _IN_ int len_V,
    _IN_ int len_E,
    _OUT_ int* d_deg
)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= len_V) {
        return;
    }
    d_deg[tid] = d_V[tid + 1] - d_V[tid];
}



static __global__ void d_k_core_scan(
    _IN_ int* d_deg,
    _IN_ int len_V,
    _IN_ int level,
    _IN_ int* d_buf_2D,
    _IN_ int* d_buf_tail_2D
)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int threads_num = blockDim.x * gridDim.x;
    int* d_buf = d_buf_2D + blockIdx.x * len_V;
    __shared__ int buf_tail;

    if (threadIdx.x == 0) {
        buf_tail = 0;
    }
    __syncthreads();

    for (int base = 0; base < len_V; base += threads_num) {
        int v = base + tid;

        if (v >= len_V) {
            continue;
        }

        if (d_deg[v] == level) {
            int buf_idx = atomicAdd(&buf_tail, 1);
            d_buf[buf_idx] = v;
        }

    }
    __syncthreads();    

    if (threadIdx.x == 0) {
        d_buf_tail_2D[blockIdx.x] = buf_tail;
    }
}



static __global__ void d_k_core_loop(
    _IN_ int* d_V,
    _IN_ int* d_E,
    _OUT_ int* d_deg,
    _IN_ int len_V,
    _IN_ int len_E,
    _IN_ int level,
    _IN_ int* d_buf_2D,
    _IN_ int* d_buf_tail_2D,
    _OUT_ int* d_count
)
{
    int warp_size = 32;
    int tid = threadIdx.x;
    int* d_buf = d_buf_2D + blockIdx.x * len_V;
    int warp_id = tid / warp_size;
    int lane_id = tid % warp_size;
    int reg_tail;
    int i;

    __shared__ int buf_tail;
    __shared__ int base;

    if (threadIdx.x == 0) {
        base = 0;
        buf_tail = d_buf_tail_2D[blockIdx.x];
    }
    __syncthreads();

    while (1) {
        __syncthreads();

        if (base == buf_tail) {
            break;
        }

        i = base + warp_id;
        reg_tail = buf_tail;
        __syncthreads();

        if (i >= reg_tail) {
            continue;
        }

        if (threadIdx.x == 0) {
            base += blockDim.x / warp_size;
            if (reg_tail < base) {
                base = reg_tail;
            }
        }

        int v = d_buf[i];
        int edge_start = d_V[v];
        int edge_end = d_V[v + 1];

        while (1) {
            __syncwarp();

            if (edge_start >= edge_end) {
                break;
            }

            int curr_e = edge_start + lane_id;
            edge_start += warp_size;

            if (curr_e >= edge_end) {
                continue;
            }

            int nbr = d_E[curr_e];
            if (d_deg[nbr] > level) {
                int a = atomicSub(d_deg + nbr, 1);

                if (a == level + 1) {
                    int loc = atomicAdd(&buf_tail, 1);
                    d_buf[loc] = nbr;
                }

                if (a <= level) {
                    atomicAdd(d_deg + nbr, 1);
                }
            }
        }
    }

    if (threadIdx.x == 0 && buf_tail) {
        atomicAdd(d_count, buf_tail);
    }
    
}


int cuda_k_core (
    _IN_ int* V,
    _IN_ int* E,
    _IN_ int len_V,
    _IN_ int len_E,
    _OUT_ int* k_core_res
)
{
    int cuda_ret = hipSuccess;
    int EG_ret = EG_GPU_SUCC;

    int block_size = 256;
    int grid_size = 56; //TODO

    int count = 0, level = 0;

    int *d_V = NULL, *d_E = NULL, *d_deg = NULL, *d_k_core_res = NULL,
            *d_buf_2D = NULL, *d_buf_tail_2D = NULL, *d_count = NULL;

    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_V, sizeof(int) * (len_V + 1)));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_E, sizeof(int) * len_E));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_deg, sizeof(int) * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_k_core_res, sizeof(int) * len_V));
    // TMP TODO TODO size
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_buf_2D, sizeof(int) * grid_size * len_V));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_buf_tail_2D, sizeof(int) * grid_size));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_count, sizeof(int)));

    EXIT_IF_CUDA_FAILED(hipMemcpy(d_V, V, sizeof(int) * (len_V + 1), hipMemcpyHostToDevice));
    EXIT_IF_CUDA_FAILED(hipMemcpy(d_E, E, sizeof(int) * len_E, hipMemcpyHostToDevice));

    EXIT_IF_CUDA_FAILED(hipMemset(d_count, 0, sizeof(int)));

    d_calc_deg<<<len_V / block_size + 1, block_size>>>(d_V, d_E, len_V, len_E, d_deg);
    while (count < len_V) {
        EXIT_IF_CUDA_FAILED(hipMemset(d_buf_tail_2D, 0, sizeof(int) * grid_size));

        d_k_core_scan<<<grid_size, block_size>>>(d_deg, len_V, level, d_buf_2D, d_buf_tail_2D);

        d_k_core_loop<<<grid_size, block_size>>>(d_V, d_E, d_deg, len_V, len_E, level,
                                                    d_buf_2D, d_buf_tail_2D, d_count);
        
        EXIT_IF_CUDA_FAILED(hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));

        ++level;
    }

    EXIT_IF_CUDA_FAILED(hipMemcpy(k_core_res, d_deg, sizeof(int) * len_V, hipMemcpyDeviceToHost));

exit:
    hipFree(d_V);
    hipFree(d_E);
    hipFree(d_deg);
    hipFree(d_k_core_res);
    hipFree(d_buf_2D);
    hipFree(d_buf_tail_2D);
    hipFree(d_count);

    if (cuda_ret != hipSuccess) {
        switch (cuda_ret) {
            case hipErrorOutOfMemory:
                EG_ret = EG_GPU_FAILED_TO_ALLOCATE_DEVICE_MEM;
                break;
            default:
                EG_ret = EG_GPU_DEVICE_ERR;
                break;
        }
    }

    return EG_ret;
}

} // namespace gpu_easygraph